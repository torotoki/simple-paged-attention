#include "hip/hip_runtime.h"
#include <cmath>
#include <cassert>
#include "common/host_utils.h"
#include "common/cuda_utility.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include "paging_manager.cu"

namespace cg = cooperative_groups;

namespace GPUAutoregressiveAttention {
__global__ void simple_gemm_kernel(
    int N, int M, int L,
    float* A, float* B, float* C
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (global_idx >= N || global_idy >= L) return;
    
    float sum = 0.0f;
    for (int j = 0; j < M; ++j) {
      sum += A[global_idx * M + j] * B[j * L + global_idy];
    }
    C[global_idx * L + global_idy] = sum;
}

void launch_simple_gemm_kernel(
  int N,
  int M,
  int L,
  float* d_A,
  float* d_B,
  float* d_out
) {
  dim3 block_size(16, 16);
  dim3 grid_size(
      (N + block_size.x - 1) / block_size.x,
      (L + block_size.y - 1) / block_size.y
  );
  simple_gemm_kernel<<<grid_size, block_size>>>(
      N, M, L,
      d_A, d_B, d_out
  );
}

__global__ void simple_gemm_with_cache_kernel(
    int N, int M, int L,
    float* A, float* B,
    unsigned int* C_page_table, float* C_blocks
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (global_idx >= N || global_idy >= L) return;
    
    float sum = 0.0f;
    for (int j = 0; j < M; ++j) {
      sum += A[global_idx * M + j] * B[j * L + global_idy];
    }
    int physical_address = translate_address(
        global_idx,
        global_idy / TOKENS_PER_BLOCK,
        C_page_table
    );

    //C[global_idx * L + global_idy] = sum;
    C_page_table[physical_address + (global_idy % TOKENS_PER_BLOCK)] = sum;
}

void launch_simple_gemm_kernel_with_cache(
  int N,
  int M,
  int L,
  float* d_A,
  float* d_B,
  unsigned int* d_out_page_table,
  float* d_out_blocks
) {
  dim3 block_size(16, 16);
  dim3 grid_size(
      (N + block_size.x - 1) / block_size.x,
      (L + block_size.y - 1) / block_size.y
  );
  simple_gemm_with_cache_kernel<<<grid_size, block_size>>>(
      N, M, L,
      d_A, d_B,
      d_out_page_table, d_out_blocks
  );
}

__global__ void transpose_gemm_imbalance_kernel(
    int N, int r, int M,
    float* A, float* B, float* out
) {
  /**
   * Compute A ・B^T
   * A: N x M
   * B: r x M
   * out: N x r
   */
  int global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  int global_idy = blockDim.y * blockIdx.y + threadIdx.y;
  if (global_idx >= r || global_idy >= N) return;

  float sum = 0.0f;
  for (int j = 0; j < M; ++j) {
    // Q (N x d_k),  K (N x d_k)
    // Q (N x d_k) \dot K^T (d_k x N)
    // out (N x N)
    sum += A[global_idy * M + j] * B[global_idx * M + j];
  }
  out[global_idy * r + global_idx] = sum;
}

void launch_transpose_gemm_imbalance_kernel(
  int N,
  int r,
  int M,
  float* A,
  float* B,
  float* out
) {
  dim3 block_size(16, 1);
  dim3 grid_size(
      (r + block_size.x - 1) / block_size.x,
      (N + block_size.y - 1) / block_size.y
  );
  transpose_gemm_imbalance_kernel<<<grid_size, block_size>>>(
      N, r, M, A, B, out
  );
}

__global__ void softmax_norm_kernel(
    int N, int M,
    float* A, float norm
) {
  /**
   * Very naive implementation of softmax
   * 1) 行ごとにsum_exp, max を集計する
   * 2) exp((A[i, j] - max) / d_k) / sum_exp
   *   sum_exp = Sigma_j exp((A[i, j] - max) / d_k)
   *
   * TODO: Separate max() and sum() computations and
   * then compute softmax computation.
   * Currently this does not support large M > (block size)
   **/
  cg::thread_block cta = cg::this_thread_block();
  int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int global_idy = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ float max_value, sum;
  max_value = A[global_idy * M + 0] / norm;
  sum = 0.0f;
  if (threadIdx.x == 0) {
    for (int j = 0; j < M; ++j) {
      max_value = max(max_value, A[global_idy * M + j] / norm);
    }
  }
  if (threadIdx.x == 0) {
    for (int j = 0; j < M; ++j) {
      sum += exp(A[global_idy * M + j] / norm - max_value);
    }
  }
  cg::sync(cta);

  if (global_idx < M && global_idy < N) {
    A[global_idy * M + global_idx] =
      exp(A[global_idy * M + global_idx] / norm - max_value) / sum;
  }
}


void launch_softmax_norm_kernel(
  int N,
  int M,
  float* A,
  int d_k = 1
) {
  assert(N > 0 && M > 0);
  float norm = std::sqrt(d_k);

  dim3 block_size(128, 1);
  dim3 grid_size(
      (M + block_size.x - 1) / block_size.x,
      (N + block_size.y - 1) / block_size.y
  );
  softmax_norm_kernel<<<grid_size, block_size>>>(
      N, M, A, norm
  );
}


Matrix<float> launch_autoregressive_attention_kernels(
    int context_size,
    int d_model,
    int d_k,
    Matrix<float>& h_W_Q,
    Matrix<float>& h_W_K,
    Matrix<float>& h_W_V,
    Matrix<float>& h_X,
    bool enable_kv_cache = false,
    bool verbose = false
) {
  // write code here

  float* d_W_Q, *d_W_K, *d_W_V, *d_X;
  size_t input_size = context_size * d_model * sizeof(float);
  size_t weight_size = d_model * d_k * sizeof(float);

  checkCudaErrors(hipMalloc(&d_W_Q, weight_size));
  checkCudaErrors(hipMalloc(&d_W_K, weight_size));
  checkCudaErrors(hipMalloc(&d_W_V, weight_size));
  checkCudaErrors(hipMalloc(&d_X, input_size));
  checkCudaErrors(hipMemcpy(d_W_Q, h_W_Q.get(), weight_size, hipMemcpyDefault));
  checkCudaErrors(hipMemcpy(d_W_K, h_W_K.get(), weight_size, hipMemcpyDefault));
  checkCudaErrors(hipMemcpy(d_W_V, h_W_V.get(), weight_size, hipMemcpyDefault));
  checkCudaErrors(hipMemcpy(d_X, h_X.get(), input_size, hipMemcpyDefault));

  // Intermediate output for projection
  // float *d_K_cache, *d_V_cache;
  // checkCudaErrors(hipMalloc(&d_K_cache, context_size * d_k * sizeof(float)));
  // checkCudaErrors(hipMalloc(&d_V_cache, context_size * d_k * sizeof(float)));

  // Initialize paging manager for KV-caches
  int *K_page_table, *V_page_table;
  int *K_blocks, *V_blocks;
  init_page_table(K_page_table);
  init_page_table(V_page_table);
  init_blocks(K_blocks, d_k);
  init_blocks(V_blocks, d_k);

  // Reuse d_Q over the time
  float* d_Q;
  checkCudaErrors(hipMalloc(&d_Q, 1 * d_k * sizeof(float)));

  // Variables for stacking outputs
  Matrix<float> h_out(context_size, d_k);
  float *d_out;
  size_t output_size = context_size * d_k * sizeof(float);
  checkCudaErrors(hipMalloc(&d_out, output_size));

  // Compute the multiplicative attention iteratively
  for (int t = 1; t <= context_size; ++t) {
    // Q: 1 x d_k
    
    // h_last_word_embed: 1 x d_model
    float* d_last_word_embed = &d_X[h_X.num_cols * (t - 1)];

    // Q (1 x d_k) = last_word_embed (1 x d_model) ・W_Q (d_model x d_k)
    launch_simple_gemm_kernel(1, d_model, d_k, d_last_word_embed, d_W_Q, d_Q);
    
    if (!enable_kv_cache) {
      // TODO: replace launch_simple_gemm_kernel with KV-cache
      launch_simple_gemm_kernel_with_cache(t, d_model, d_k, d_X, d_W_K, K_blocks, K_page_table);
      launch_simple_gemm_kernel_with_cache(t, d_model, d_k, d_X, d_W_V, V_blocks, V_page_table);
    } else {
      // Reuse parts of K and V that have already been computed
      if (t == 1) {
        launch_simple_gemm_kernel_with_cache(1, d_model, d_k, d_X, d_W_K, K_blocks, K_page_table);
        launch_simple_gemm_kernel_with_cache(1, d_model, d_k, d_X, d_W_V, V_blocks, V_page_table);
      } else {
        float* d_new_embed = d_X + (t - 1) * d_model;
        float* d_K_tail = d_K_cache + (t - 1) * d_k;
        float* d_V_tail = d_V_cache + (t - 1) * d_k;
        launch_simple_gemm_kernel_with_cache(1, d_model, d_k, d_new_embed, d_W_K, d_K_tail);
        launch_simple_gemm_kernel_with_cache(1, d_model, d_k, d_new_embed, d_W_V, d_V_tail);
      }
    }

    // QKT (1 x t) = Q (1 x d_k) ・K^T (t x d_k)^T
    float* d_QKT;
    checkCudaErrors(hipMalloc(&d_QKT, 1 * t * sizeof(float)));
    launch_transpose_gemm_imbalance_kernel(1, t, d_k, d_Q, d_K_cache, d_QKT);
    
    launch_softmax_norm_kernel(1, t, d_QKT, d_k);
    
    // Extract t-th output embedding
    float* d_out_single_word = &d_out[h_out.num_cols * (t - 1)];
    launch_simple_gemm_kernel(1, t, d_k, d_QKT, d_V_cache, d_out_single_word);

    checkCudaErrors(hipFree(d_QKT));
  }

  checkCudaErrors(hipMemcpy(h_out.get(), d_out, output_size, hipMemcpyDefault));
  hipDeviceSynchronize();

  // Free memory
  checkCudaErrors(hipFree(d_out));
  checkCudaErrors(hipFree(d_Q));
  checkCudaErrors(hipFree(d_K_cache));
  checkCudaErrors(hipFree(d_V_cache));
  checkCudaErrors(hipFree(d_W_Q));
  checkCudaErrors(hipFree(d_W_K));
  checkCudaErrors(hipFree(d_W_V));
  checkCudaErrors(hipFree(d_X));

  return h_out;
}
}
