#include "hip/hip_runtime.h"
#include <cmath>
#include <cassert>
#include "common/host_utils.h"
#include "common/cuda_utility.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include "paging_manager.cu"

namespace cg = cooperative_groups;

namespace PagedAttention {

__global__ void simple_gemm_kernel(
    int N, int M, int L,
    float* A, float* B, float* C
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (global_idx >= N || global_idy >= L) return;
    
    float sum = 0.0f;
    for (int j = 0; j < M; ++j) {
      sum += A[global_idx * M + j] * B[j * L + global_idy];
    }
    C[global_idx * L + global_idy] = sum;
}

void launch_simple_gemm_kernel(
  int N,
  int M,
  int L,
  float* d_A,
  float* d_B,
  float* d_out
) {
  dim3 block_size(16, 16);
  dim3 grid_size(
      (N + block_size.x - 1) / block_size.x,
      (L + block_size.y - 1) / block_size.y
  );
  simple_gemm_kernel<<<grid_size, block_size>>>(
      N, M, L,
      d_A, d_B, d_out
  );
}

__global__ void simple_gemm_kernel(
    int N, int M, int L,
    float* A, Blocks<float>* B_blocks, float* C
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (global_idx >= N || global_idy >= L) return;
    
    float sum = 0.0f;
    for (int j = 0; j < M; ++j) {
      float B_elem = *fetch_block(j, global_idy, B_blocks);
      sum += A[global_idx * M + j] * B_elem;
    }
    C[global_idx * L + global_idy] = sum;
}

void launch_simple_gemm_kernel(
  int N,
  int M,
  int L,
  float* d_A,
  Blocks<float>* B_blocks,
  float* d_out
) {
  dim3 block_size(16, 16);
  dim3 grid_size(
      (N + block_size.x - 1) / block_size.x,
      (L + block_size.y - 1) / block_size.y
  );
  simple_gemm_kernel<<<grid_size, block_size>>>(
      N, M, L,
      d_A, B_blocks, d_out
  );
}

__global__ void simple_gemm_with_cache_kernel(
    int N, int M, int L,
    float* A, float* B,
    Blocks<float>* C_blocks
) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (global_idx >= N || global_idy >= L) return;
    
    float sum = 0.0f;
    for (int j = 0; j < M; ++j) {
      sum += A[global_idx * M + j] * B[j * L + global_idy];
    }
    float* out_addr = fetch_block<float>(
        global_idx, // seq_idx
        global_idy,  // token_idx
        C_blocks
    );
    
    *out_addr = sum;
}

void launch_simple_gemm_kernel_with_cache(
  int N,
  int M,
  int L,
  float* d_A,
  float* d_B,
  Blocks<float>* d_out_blocks
) {
  dim3 block_size(16, 16);
  dim3 grid_size(
      (N + block_size.x - 1) / block_size.x,
      (L + block_size.y - 1) / block_size.y
  );
  simple_gemm_with_cache_kernel<<<grid_size, block_size>>>(
      N, M, L,
      d_A, d_B,
      d_out_blocks
  );
}

__global__ void transpose_gemm_imbalance_kernel(
    int N, int r, int M,
    float* A, Blocks<float>* B_blocks, float* out
) {
  /**
   * Compute A ・B^T
   * A: N x M
   * B: r x M
   * out: N x r
   */
  int global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  int global_idy = blockDim.y * blockIdx.y + threadIdx.y;
  if (global_idx >= r || global_idy >= N) return;

  float sum = 0.0f;
  for (int j = 0; j < M; ++j) {
    // Q (N x d_k),  K (N x d_k)
    // Q (N x d_k) \dot K^T (d_k x N)
    // out (N x N)
    float B_value = *fetch_block(global_idx, j, B_blocks);
    sum += A[global_idy * M + j] * B_value;
  }
  out[global_idy * r + global_idx] = sum;
}

void launch_transpose_gemm_imbalance_kernel(
  int N,
  int r,
  int M,
  float* A,
  Blocks<float>* B_blocks,
  float* out
) {
  dim3 block_size(16, 1);
  dim3 grid_size(
      (r + block_size.x - 1) / block_size.x,
      (N + block_size.y - 1) / block_size.y
  );
  transpose_gemm_imbalance_kernel<<<grid_size, block_size>>>(
      N, r, M, A, B_blocks, out
  );
}

__global__ void softmax_norm_kernel(
    int N, int M,
    float* A, float norm
) {
  /**
   * Very naive implementation of softmax
   * 1) 行ごとにsum_exp, max を集計する
   * 2) exp((A[i, j] - max) / d_k) / sum_exp
   *   sum_exp = Sigma_j exp((A[i, j] - max) / d_k)
   *
   * TODO: Separate max() and sum() computations and
   * then compute softmax computation.
   * Currently this does not support large M > (block size)
   **/
  cg::thread_block cta = cg::this_thread_block();
  int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int global_idy = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ float max_value, sum;
  max_value = A[global_idy * M + 0] / norm;
  sum = 0.0f;
  if (threadIdx.x == 0) {
    for (int j = 0; j < M; ++j) {
      max_value = max(max_value, A[global_idy * M + j] / norm);
    }
  }
  if (threadIdx.x == 0) {
    for (int j = 0; j < M; ++j) {
      sum += exp(A[global_idy * M + j] / norm - max_value);
    }
  }
  cg::sync(cta);

  if (global_idx < M && global_idy < N) {
    A[global_idy * M + global_idx] =
      exp(A[global_idy * M + global_idx] / norm - max_value) / sum;
  }
}


void launch_softmax_norm_kernel(
  int N,
  int M,
  float* A,
  int d_k = 1
) {
  assert(N > 0 && M > 0);
  float norm = std::sqrt(d_k);

  dim3 block_size(128, 1);
  dim3 grid_size(
      (M + block_size.x - 1) / block_size.x,
      (N + block_size.y - 1) / block_size.y
  );
  softmax_norm_kernel<<<grid_size, block_size>>>(
      N, M, A, norm
  );
}


Matrix<float> launch_paged_attention_kernels(
    int context_size,
    int d_model,
    int d_k,
    Matrix<float>& h_W_Q,
    Matrix<float>& h_W_K,
    Matrix<float>& h_W_V,
    Matrix<float>& h_X,
    bool enable_kv_cache = false,
    bool verbose = false
) {
  // write code here

  float* d_W_Q, *d_W_K, *d_W_V, *d_X;
  size_t input_size = context_size * d_model * sizeof(float);
  size_t weight_size = d_model * d_k * sizeof(float);

  checkCudaErrors(hipMalloc(&d_W_Q, weight_size));
  checkCudaErrors(hipMalloc(&d_W_K, weight_size));
  checkCudaErrors(hipMalloc(&d_W_V, weight_size));
  checkCudaErrors(hipMalloc(&d_X, input_size));
  checkCudaErrors(hipMemcpy(d_W_Q, h_W_Q.get(), weight_size, hipMemcpyDefault));
  checkCudaErrors(hipMemcpy(d_W_K, h_W_K.get(), weight_size, hipMemcpyDefault));
  checkCudaErrors(hipMemcpy(d_W_V, h_W_V.get(), weight_size, hipMemcpyDefault));
  checkCudaErrors(hipMemcpy(d_X, h_X.get(), input_size, hipMemcpyDefault));

  // Initialize paging manager on Unified Memory for KV-caches
  Blocks<float>* K_blocks = nullptr;
  Blocks<float>* V_blocks = nullptr;
  checkCudaErrors(hipMallocManaged(&K_blocks, sizeof(Blocks<float>)));
  checkCudaErrors(hipMallocManaged(&V_blocks, sizeof(Blocks<float>)));
  K_blocks->block_table = nullptr;
  K_blocks->blocks = nullptr;
  V_blocks->block_table = nullptr;
  V_blocks->blocks = nullptr;
  // checkCudaErrors(hipMalloc(&K_blocks->block_table, 2048 * sizeof(int)));
  // hipGetLastError();
  // printf("Malloc successed!\n");
  init_page_table(K_blocks->block_table);
  init_page_table(V_blocks->block_table);
  init_blocks(K_blocks->blocks, d_k);
  init_blocks(V_blocks->blocks, d_k);

  // Reuse d_Q over the time
  float* d_Q;
  checkCudaErrors(hipMalloc(&d_Q, 1 * d_k * sizeof(float)));

  // Variables for stacking outputs
  Matrix<float> h_out(context_size, d_k);
  float *d_out;
  size_t output_size = context_size * d_k * sizeof(float);
  checkCudaErrors(hipMalloc(&d_out, output_size));

  // TODO: allocate memory for paging mechanisms: block_table and blocks

  // Compute the multiplicative attention iteratively
  for (int t = 1; t <= context_size; ++t) {
    // Q: 1 x d_k
    
    // h_last_word_embed: 1 x d_model
    float* d_last_word_embed = &d_X[h_X.num_cols * (t - 1)];

    // Q (1 x d_k) = last_word_embed (1 x d_model) ・W_Q (d_model x d_k)
    launch_simple_gemm_kernel(1, d_model, d_k, d_last_word_embed, d_W_Q, d_Q);
    
    if (!enable_kv_cache) {
      launch_simple_gemm_kernel_with_cache(t, d_model, d_k, d_X, d_W_K, K_blocks);
      launch_simple_gemm_kernel_with_cache(t, d_model, d_k, d_X, d_W_V, V_blocks);
    } else {
      // Reuse parts of K and V that have already been computed
      // TODO: Add methods to force to update the partial results of GEMM
      // if (t == 1) {
      //   launch_simple_gemm_kernel_with_cache(1, d_model, d_k, d_X, d_W_K, K_blocks, K_page_table);
      //   launch_simple_gemm_kernel_with_cache(1, d_model, d_k, d_X, d_W_V, V_blocks, V_page_table);
      // } else {
      //   float* d_new_embed = d_X + (t - 1) * d_model;
      //   float* d_K_tail = d_K_cache + (t - 1) * d_k;
      //   float* d_V_tail = d_V_cache + (t - 1) * d_k;
      //   launch_simple_gemm_kernel_with_cache(1, d_model, d_k, d_new_embed, d_W_K, d_K_tail);
      //   launch_simple_gemm_kernel_with_cache(1, d_model, d_k, d_new_embed, d_W_V, d_V_tail);
      // }
    }

    // QKT (1 x t) = Q (1 x d_k) ・K^T (t x d_k)^T
    float* d_QKT;
    checkCudaErrors(hipMalloc(&d_QKT, 1 * t * sizeof(float)));
    launch_transpose_gemm_imbalance_kernel(1, t, d_k, d_Q, K_blocks, d_QKT);
    
    launch_softmax_norm_kernel(1, t, d_QKT, d_k);
    
    // Extract t-th output embedding
    float* d_out_single_word = &d_out[h_out.num_cols * (t - 1)];
    launch_simple_gemm_kernel(1, t, d_k, d_QKT, V_blocks, d_out_single_word);

    checkCudaErrors(hipFree(d_QKT));
  }

  checkCudaErrors(hipMemcpy(h_out.get(), d_out, output_size, hipMemcpyDefault));
  hipDeviceSynchronize();

  // Free memory
  checkCudaErrors(hipFree(d_out));
  checkCudaErrors(hipFree(d_Q));
  // checkCudaErrors(hipFree(d_K_cache));
  // checkCudaErrors(hipFree(d_V_cache));
  checkCudaErrors(hipFree(d_W_Q));
  checkCudaErrors(hipFree(d_W_K));
  checkCudaErrors(hipFree(d_W_V));
  checkCudaErrors(hipFree(d_X));

  return h_out;
}
}
